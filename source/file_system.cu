#include "hip/hip_runtime.h"
﻿
#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 create_time = 0;
__device__ __managed__ u32 modified_time = 0;

//init volumn superblock
__device__ void init_superblock(FileSystem *fs) {
  for (int i = 0; i < fs->SUPERBLOCK_SIZE; i++) {
    fs->volume[i] = 0;
  } 
}

//init file-control block 32kb
__device__ void init_FCB(FileSystem *fs) {
  for (int i = 0; i < fs->FCB_ENTRIES; i++) {
    fs->volume[i * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE] = '\0';
  }
}


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
  fs->FILE_ADDING_ADDRESS = FILE_BASE_ADDRESS;


  //init 
  init_superblock(fs);

  //init
  init_FCB(fs);
}


__device__ bool compare_file_name(char *filename, char *searchname) {
  bool check = true;
  for (int i = 0; i < 20; i++) {
    if (*filename != *searchname) {
      check = false;
      break;
    }
    filename++;
    searchname++;
  }
  return check;
}

__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */ 
  
  u32 file_pointer = -1;
  for (int i = 0; i < fs->FCB_ENTRIES; i++) {
    u32 addr_entry = i * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
    if (fs->volume[addr_entry] != '\0') {
      bool check_similarity = compare_file_name(s, (char*) &fs->volume[addr_entry]);
      if (check_similarity) {
        file_pointer = i;
        break;
      } 
    } 
  }

  if (file_pointer != -1) {
    //update modified time
    modified_time++;
    fs->volume[file_pointer * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE + 22] = modified_time / 256;
    fs->volume[file_pointer * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE + 23] = modified_time % 256;
    return file_pointer;
  } else {
    if (op == G_WRITE) {
      for (int i = 0; i < fs->FCB_ENTRIES; i++) {
        int temp_entry = i * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
        if (fs->volume[temp_entry] == '\0') {
          file_pointer = i;
          break;
        }
      }

      int length = 0;
      int file_entry = file_pointer * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
      while (s[length] != '\0') {
        fs->volume[file_entry + length] = s[length];
        length++;
        if (length == fs->MAX_FILENAME_SIZE) {
          printf("filename length exceeds max size.");
          break;
        }
      }
      fs->volume[file_entry + length] = '\0';

      /*
      index: 
      0-19 filename
      20-21 create_time
      22-23 modified_time
      24-27 size
      28-29 address
      */

      //set create_time
      fs->volume[file_entry + 20] = create_time / 256;
      fs->volume[file_entry + 21] = create_time % 256;
      create_time++;
  
      //set modified_time
      fs->volume[file_entry + 22] = modified_time / 256;
      fs->volume[file_entry + 23] = modified_time % 256;
      modified_time++;
      
      //set size
      u32 size = 0;
      fs->volume[file_entry + 24] = size % 256;
      fs->volume[file_entry + 25] = (size>>8) % 256;
      fs->volume[file_entry + 26] = (size>>16) % 256;
      fs->volume[file_entry + 27] = (size>>24) % 256;
      //set address
      fs->volume[file_entry + 28] = fs->FILE_ADDING_ADDRESS / 256;
      fs->volume[file_entry + 29] = fs->FILE_ADDING_ADDRESS % 256;
      
      return file_pointer;
    } 
  } 
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */ 
  if (fp != -1 && fs->volume[fp * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE] != '\0') {
    u32 addr_entry = fs->volume[fp * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE + 28] * 256 + fs->volume[fp * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE + 29];
    for (int i = 0; i < size; i++) {
      output[i] = fs->volume[addr_entry + i];
    }
  }
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	/* Implement write operation here */  
  u32 addr_entry = fp * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
  u32 file_pointer = fs->volume[addr_entry + 28] * 256 + fs->volume[addr_entry + 29];
  u32 old_size = fs->volume[addr_entry + 24] + fs->volume[addr_entry + 25]<<8 + fs->volume[addr_entry + 26]<<16 + fs->volume[addr_entry + 27]<<24;
  //set new size
  fs->volume[addr_entry + 24] = size % 256;
  fs->volume[addr_entry + 25] = (size>>8) % 256;
  fs->volume[addr_entry + 26] = (size>>16) % 256;
  fs->volume[addr_entry + 27] = (size>>24) % 256;

  //file is empty, directly write in data
  if (fs->FILE_ADDING_ADDRESS == fs->FILE_BASE_ADDRESS) {
    for (int i = 0; i < size; i++) {
      fs->volume[file_pointer + i] = input[i];
    }
    fs->FILE_ADDING_ADDRESS += size;
  } else if (fs->FILE_ADDING_ADDRESS == fp * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE + old_size) {
    for (int i = 0; i < size; i++) {
      fs->volume[file_pointer + i] = input[i];
    }
    if (size < old_size) {
      for (int i = file_pointer + old_size-size; i < file_pointer + old_size; i++) {
        fs->volume[i] = '\0';
      }
    } 
    
    fs->FILE_ADDING_ADDRESS = fs->FILE_ADDING_ADDRESS-old_size + size;
  } else {
    u32 new_addr = fs->FILE_ADDING_ADDRESS-old_size;
    u32 i = file_pointer;
    while (i < new_addr) {
      fs->volume[i] = fs->volume[i + old_size];
      i++;
    }
    for (int j = 0; j < size; j++) {
      fs->volume[new_addr + j] = input[j];
    }

    if (size < old_size) {
      for (int k = 0; k < old_size-size; k++) {
        fs->volume[fs->FILE_ADDING_ADDRESS-k] = '\0';
      }
    }
    for (int i = 0; i < fs->FCB_ENTRIES; i++) {
      u32 FCB_start = i * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
      u32 addr = fs->volume[FCB_start + 28] * 256 + fs->volume[FCB_start + 29]-old_size;
      if (i != addr_entry && addr >= file_pointer) {
        if (fs->volume[FCB_start] != '\0') {
          fs->volume[FCB_start + 28] = addr / 256;
          fs->volume[FCB_start + 29] = addr % 256;
        }
      }
    }
    fs->volume[addr_entry + 28] = new_addr / 256;
    fs->volume[addr_entry + 29] = new_addr % 256;

    fs->FILE_ADDING_ADDRESS = new_addr + size;
  }
  
}

__device__ void fs_gsys(FileSystem *fs, int op) 
{
	/* Implement LS_D and LS_S operation here */  
  if (op == LS_D) {
    printf("===sort by modified time===\n");
    int temp_fcb[32];
    for (int i = 1; i < fs->FCB_ENTRIES; i++) {
      u32 addr_entry = i * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
      if (fs->volume[addr_entry] != '\0') {
        int current_modified_time = fs->volume[addr_entry + 22] * 256 + fs->volume[addr_entry + 23];
        int index = i;
        for (int j = i-1; j >= 0; j--) {
          u32 addr_entry2 = j * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
          if (fs->volume[addr_entry2] != '\0') {
            int previous_modified_time = fs->volume[addr_entry2 + 22] * 256 + fs->volume[addr_entry2 + 23];
            if (previous_modified_time < current_modified_time) {
              for (int k = 0; k < fs->FCB_SIZE; k++) {
                temp_fcb[k] = fs->volume[index * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE + k];
                fs->volume[index * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE + k] = fs->volume[addr_entry2 + k];
                fs->volume[addr_entry2 + k] = temp_fcb[k];
              }
              index--;
            } 
          }
        }
      }
    }

    for (int i = 0; i < fs->FCB_ENTRIES; i++) {
      u32 addr_entry = i * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
      if (fs->volume[addr_entry] != '\0') {
        char* head = (char*) &fs->volume[addr_entry];
        printf("%s", head);        
        printf("\n");
      }
    }
    
  } else if (op == LS_S) {
    printf("===sort by file size===\n");
    int temp_fcb[32];
    for (int i = 1; i < fs->FCB_ENTRIES; i++) {
      u32 addr_entry = i * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
      if (fs->volume[addr_entry] != '\0') {
        int current_file_size = fs->volume[addr_entry + 24] + (fs->volume[addr_entry + 25]<<8) + (fs->volume[addr_entry + 26]<<16) + (fs->volume[addr_entry + 27]<<24);
        int current_create_time = fs->volume[addr_entry + 20] * 256 + fs->volume[addr_entry + 21];
        int index = i;
        for (int j = i-1; j >= 0; j--) {
          u32 addr_entry2 = j * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
          if (fs->volume[addr_entry2] != '\0') {
            int previous_file_size = fs->volume[addr_entry2 + 24] + (fs->volume[addr_entry2 + 25]<<8) + (fs->volume[addr_entry2 + 26]<<16) + (fs->volume[addr_entry2 + 27]<<24);
            int previous_create_time = fs->volume[addr_entry2 + 20] * 256 + fs->volume[addr_entry2 + 21];
            if (previous_file_size < current_file_size || (previous_file_size == current_file_size && current_create_time < previous_create_time)) {
              for (int k = 0; k < fs->FCB_SIZE; k++) {
                temp_fcb[k] = fs->volume[index * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE + k];
                fs->volume[index * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE + k] = fs->volume[addr_entry2 + k];
                fs->volume[addr_entry2 + k] = temp_fcb[k];
              }
              index--;
            }
          }
        }
      }
    }
    
    for (int i = 0; i < fs->FCB_ENTRIES; i++) {
      u32 addr_entry = i * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
      if (fs->volume[addr_entry] != '\0') {
        char* head = (char*) &fs->volume[addr_entry];
        u32 file_size = fs->volume[addr_entry + 24] + (fs->volume[addr_entry + 25]<<8) + (fs->volume[addr_entry + 26]<<16) + (fs->volume[addr_entry + 27]<<24);
        printf("%s", head);
        printf(" %d\n", file_size);
      }
    }
  }
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */ 
  if (op == RM) {
    //check whether the file exists
    u32 file_pointer = -1;
    for (int i = 0; i < fs->FCB_ENTRIES; i++) {
      u32 addr_entry = i * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
      if (fs->volume[addr_entry] != '\0') {
        bool check_similarity = compare_file_name(s, (char*) &fs->volume[addr_entry]);
        if (check_similarity) {
          file_pointer = i;
          break;
        } 
      } 
    }

    u32 addr_entry = file_pointer * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
    if (file_pointer == -1) {
      printf("File %s does not exist!\n", s);
    } else {
      //clean the content
      u32 addr = fs->volume[addr_entry + 28] * 256 + fs->volume[addr_entry + 29];
      u32 file_size = fs->volume[addr_entry + 24] + fs->volume[addr_entry + 25]<<8 + fs->volume[addr_entry + 26]<<16 + fs->volume[addr_entry + 27]<<24;
      for (u32 i = addr; i < fs->FILE_ADDING_ADDRESS; i++) {
        if (i < fs->FILE_ADDING_ADDRESS-file_size) {
          fs->volume[i] = fs->volume[i + file_size];
        } else if (i >= fs->FILE_ADDING_ADDRESS-file_size && i < fs->FILE_ADDING_ADDRESS) {
          fs->volume[i] = '\0';
        }
      }

      fs->FILE_ADDING_ADDRESS -= file_size;
      for (int k = 0; k < fs->FCB_ENTRIES; k++) {
        u32 addr_entry2 = k * fs->FCB_SIZE + fs->SUPERBLOCK_SIZE;
        u32 new_addr = fs->volume[addr_entry2 + 28] * 256 + fs->volume[addr_entry2 + 29];
        if (fs->volume[addr_entry2] != '\0' && addr_entry2 != addr_entry) {
          if (new_addr-addr >= file_size) {
            fs->volume[addr_entry2 + 28] = (new_addr-file_size) / 256;
            fs->volume[addr_entry2 + 29] = (new_addr-file_size) % 256;
          }
        }
      }
      for (int i = 0; i < fs->FCB_SIZE; i++) {
        fs->volume[addr_entry + i] = '\0';
      }
    }
  } else {
    printf("No such operation %d\n", op);
  }
}

